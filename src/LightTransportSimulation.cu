﻿#include <iostream>

#include "rendering/Camera.cuh"
#include "rendering/Spectrum.cuh"
#include "sampling/Sampler.cuh"
#include "core/SceneLoader.cuh"
#include "lights/VisibilityTester.cuh"
#include "intergrators/PathTracingIntegrator.cuh"
#include "intergrators/BidirectionalPathIntegrator.cuh"

using namespace lts;

/*
* todo:
*
*	add comments to a bunch of files to make them more readable
*/

const static int RENDER_PIXEL_WIDTH = 1920;
const static int RENDER_PIXEL_HEIGHT = 1080;
const static int SAMPLE_PER_PIXEL = 1;
const static IntegratorType RENDERING_STRATEGY = IntegratorType::PathTracing;
const static int MAX_BOUNCE = 8;
const static int ROULETTE_START = 2;
const static bool PPM_FORMAT = true;

std::string outputFileWithoutExtension("outputs\\test");
const static std::string OUTPUT_FILE = outputFileWithoutExtension + (PPM_FORMAT ? ".ppm" : "pfm");

const char* SUBJECT_FILE = "res/holder";
const char* SCENE_FILE = "res/platforms/bigBox";

int main() {

	int version;
	hipRuntimeGetVersion(&version);
	printf("---Used CUDA version: %i---\n", version);

	hipDeviceSetLimit(hipLimitStackSize, 8192);

	std::cout << "===RENDERING PARAMETERS===\n" <<
		"	Technique: " << toString(RENDERING_STRATEGY) << "\n" <<
		"	Image resolution: " << RENDER_PIXEL_WIDTH << "x" << RENDER_PIXEL_HEIGHT << "\n" <<
		"	Samples per pixel: " << SAMPLE_PER_PIXEL << "\n" <<
		"	Maximum bounces: " << MAX_BOUNCE << std::endl;

	// Camera initialisation
	Filter* f = new GaussianFilter(Vector2f(1.0f, 1.0f), 1.0f);
	Camera* h_cam;

	// Scene initialisation
	auto start = std::chrono::high_resolution_clock::now();
	Scene* scene = parseScene(&h_cam, f,
		0.f, RENDER_PIXEL_WIDTH, RENDER_PIXEL_HEIGHT,
		SCENE_FILE, SUBJECT_FILE);
	delete f;
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
	std::cout << "(1) Scene creation finished in " <<
		duration.count() / 1000.0f << " seconds." << std::endl;

	// Light distribution initialisation
	Distribution1D* d_distribution;
	gpuErrCheck(hipMalloc((void**)&d_distribution, sizeof(Distribution1D)));
	lightDistributionKernel << <1, 1 >> > (d_distribution, scene);
	gpuErrCheck(hipDeviceSynchronize());
	gpuErrCheck(hipPeekAtLastError());

	// Thread count parameters
	dim3 block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid = dim3(RENDER_PIXEL_WIDTH / BLOCK_SIZE + (RENDER_PIXEL_WIDTH % BLOCK_SIZE != 0),
		RENDER_PIXEL_HEIGHT / BLOCK_SIZE + (RENDER_PIXEL_HEIGHT % BLOCK_SIZE != 0));

	std::cout << "	Rendering started with grid of " << grid.x << "x" << grid.y << " with block of "
		<< block.x << "x" << block.y << std::endl;

	if (RENDERING_STRATEGY == IntegratorType::PathTracing) {
		// Sampler initialisation
		Sampler h_samp = Sampler(SAMPLE_PER_PIXEL, RENDER_PIXEL_WIDTH * RENDER_PIXEL_HEIGHT);
		Sampler* d_samp = passToDevice(&h_samp);

		// Initialise integrator
		PathTracingIntegrator h_integrator(MAX_BOUNCE, ROULETTE_START, h_cam, d_samp, scene);
		PathTracingIntegrator* d_integrator = passToDevice(&h_integrator);

		// Rendering
		start = std::chrono::high_resolution_clock::now();
		PathTracingKernel << <grid, block >> > (d_integrator, d_distribution, time(0));
		gpuErrCheck(hipDeviceSynchronize());
		gpuErrCheck(hipPeekAtLastError());
		end = std::chrono::high_resolution_clock::now();
		duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
		std::cout << "(2) Ray tracing finished in " <<
			duration.count() / 1000.0f << " seconds." << std::endl;

		// Saving to image file
		h_integrator.outputResultAndOpen(OUTPUT_FILE, !PPM_FORMAT);
	}

	/**
	* Bidirectional path tracing is not functional for now:
	* Memory usage is a lot more than standard path tracing and the available memory
	* cant handle the needs
	*/
	else if (RENDERING_STRATEGY == IntegratorType::BiderectionalPathTracing) {

		// Sampler initialisation
		Sampler h_samp = Sampler(SAMPLE_PER_PIXEL, RENDER_PIXEL_WIDTH * RENDER_PIXEL_HEIGHT);
		Sampler* d_samp = passToDevice(&h_samp);

		// Initialise integrator
		BidirectionalPathIntegrator h_integrator(MAX_BOUNCE, ROULETTE_START, h_cam, d_samp, scene);
		BidirectionalPathIntegrator* d_integrator = passToDevice(&h_integrator);

		// Rendering
		start = std::chrono::high_resolution_clock::now();
		BidirectionalPathTracingKernel << <grid, block >> > (d_integrator, d_distribution, time(0));
		gpuErrCheck(hipDeviceSynchronize());
		gpuErrCheck(hipPeekAtLastError());
		end = std::chrono::high_resolution_clock::now();
		duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
		std::cout << "(2) Ray tracing finished in " <<
			duration.count() / 1000.0f << " seconds." << std::endl;

		// Saving to image file
		h_integrator.outputResultAndOpen(OUTPUT_FILE, !PPM_FORMAT);
	}

	return 0;
}